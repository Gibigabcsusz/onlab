
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <chrono>


using namespace std;
using namespace std::chrono;

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float a, float b, float *x, float *y, float *z)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
        for (int i = index; i < n; i += stride)
            z[i] = a*x[i] + b*y[i];
}

__global__
void ciklikus(float cellaSzam, int reszecskeSzam, float* helyek)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for(int i = index; i < reszecskeSzam; i += stride)
    {
        if(helyek[i] < -0.5)
            helyek[i] += cellaSzam;
        if(helyek[i] > cellaSzam-0.5)
            helyek[i] -= cellaSzam;
    }
}

int main(void)
{
    int N = 1<<5;
    float *x, *y;


    // Allocate Unified Memory -- accessible from CPU or GPU
    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // initialize x and y arrays on the host
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 5.0f*i-N;
    }

    // Launch kernel on 1M elements on the GPU
    //int blockSize = 32;

    int blockSize = 32;
    int numBlocks = (N + blockSize - 1) / blockSize;

    auto startt = high_resolution_clock::now();

    //add<<<numBlocks, blockSize>>>(N, 1.0f, 1.0f, x, y, y);

    ciklikus<<<numBlocks, blockSize >>>(N, N, y);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    auto stopp = high_resolution_clock::now();

    auto duration = duration_cast<nanoseconds>(stopp - startt);
    int nanosecs = duration.count()%1000;
    int microsecs = ((duration.count()-nanosecs)/1000)%1000;
    int millisecs = ((duration.count()-nanosecs)/1000-microsecs)/1000;
    cout << "Chrono meres: " << millisecs << " ms  +  " << microsecs << " us  +  " << nanosecs << " ns" << endl;

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        cout << i << " - " << y[i] << endl;
    //    maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;


    // Free memory
    hipFree(x);
    hipFree(y);

    return 0;
}
